#include "hip/hip_runtime.h"
// filepath: cuda-transformer/cuda-transformer/src/layers/feed_forward.cu
#include "feed_forward.cuh"
#include "utils/cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>

__global__ void feedForwardKernel(
    const float* input, float* output,
    const float* W1, const float* b1,
    const float* W2, const float* b2,
    int rows, int input_dim, int d_ff, int output_dim)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows) {
        // First layer: Linear + ReLU
        for (int j = 0; j < d_ff; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < input_dim; ++k) {
                sum += input[idx * input_dim + k] * W1[k * d_ff + j];
            }
            sum += b1[j];
            output[idx * d_ff + j] = fmaxf(0.0f, sum); // ReLU
        }

        // Second layer: Linear
        for (int j = 0; j < output_dim; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < d_ff; ++k) {
                sum += output[idx * d_ff + k] * W2[k * output_dim + j];
            }
            sum += b2[j];
            output[idx * output_dim + j] = sum; // No activation
        }
    }
}

Matrix FeedForward::forward(const Matrix &input) {
    int rows = input.getRows();
    int input_dim = input.getCols();
    int d_ff = this->d_ff;
    int output_dim = this->d_model;

    Matrix output(rows, output_dim);

    // Asume que W1, W2, b1, b2 están en memoria de dispositivo
    int blockSize = 256;
    int numBlocks = (rows + blockSize - 1) / blockSize;

    feedForwardKernel<<<numBlocks, blockSize>>>(
        input.getData(), output.getData(),
        W1.getData(), b1,
        W2.getData(), b2,
        rows, input_dim, d_ff, output_dim
    );
    hipDeviceSynchronize();

    return output;
}