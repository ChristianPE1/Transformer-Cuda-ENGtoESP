#include "hip/hip_runtime.h"
// filepath: /cuda-transformer/cuda-transformer/src/transformer/attention.cu
#include "attention.cuh"
#include "cuda_utils.cuh"
#include <cmath>

__device__ void softmax(float* data, int length) {
    float max_val = data[0];
    for (int i = 1; i < length; ++i) {
        if (data[i] > max_val) {
            max_val = data[i];
        }
    }

    float sum = 0.0f;
    for (int i = 0; i < length; ++i) {
        data[i] = exp(data[i] - max_val);
        sum += data[i];
    }

    for (int i = 0; i < length; ++i) {
        data[i] /= sum;
    }
}

__global__ void multiHeadAttentionKernel(float* queries, float* keys, float* values, 
                                         float* output, int d_model, int n_heads, 
                                         int seq_length) {
    int head_size = d_model / n_heads;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < seq_length) {
        float attention_scores[seq_length];
        for (int i = 0; i < seq_length; ++i) {
            attention_scores[i] = 0.0f;
            for (int j = 0; j < head_size; ++j) {
                attention_scores[i] += queries[idx * d_model + j] * keys[i * d_model + j];
            }
        }

        softmax(attention_scores, seq_length);

        for (int i = 0; i < seq_length; ++i) {
            output[idx * d_model + i] = 0.0f;
            for (int j = 0; j < head_size; ++j) {
                output[idx * d_model + i] += attention_scores[j] * values[i * d_model + j];
            }
        }
    }
}

void MultiHeadAttention::forward(float* queries, float* keys, float* values, 
                                  float* output, int seq_length) {
    int blockSize = 256;
    int numBlocks = (seq_length + blockSize - 1) / blockSize;
    multiHeadAttentionKernel<<<numBlocks, blockSize>>>(queries, keys, values, output, 
                                                       d_model, n_heads, seq_length);
    hipDeviceSynchronize();
}