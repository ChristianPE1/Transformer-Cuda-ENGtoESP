#include "hip/hip_runtime.h"
#include "transformer.cuh"
#include "embeddings.cuh"
#include "../utils/matrix.cuh"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <algorithm>
#include <cstdlib>
#include <ctime>

Transformer::Transformer(size_t input_vocab_size, size_t target_vocab_size,size_t d_model, size_t n_heads, size_t n_layers, size_t d_ff)
    : input_vocab_size(input_vocab_size), target_vocab_size(target_vocab_size),
      d_model(d_model), n_layers(n_layers), n_heads(n_heads), d_ff(d_ff),
      input_embedding(input_vocab_size, d_model),
      target_embedding(target_vocab_size, d_model),
      pos_encoding(d_model)
{
    // Initialize multi-layer components
    encoder_self_attention.reserve(n_layers);
    encoder_ffn.reserve(n_layers);
    decoder_self_attention.reserve(n_layers);
    decoder_cross_attention.reserve(n_layers);
    decoder_ffn.reserve(n_layers);
    
    for (size_t i = 0; i < n_layers; ++i) {
        encoder_self_attention.push_back(new MultiHeadAttention(d_model, n_heads));
        encoder_ffn.push_back(new FeedForward(d_model, d_ff));
        decoder_self_attention.push_back(new MultiHeadAttention(d_model, n_heads));
        decoder_cross_attention.push_back(new MultiHeadAttention(d_model, n_heads));
        decoder_ffn.push_back(new FeedForward(d_model, d_ff));
    }

    std::cout << "Transformer initialized with FULL ARCHITECTURE:" << std::endl;
    std::cout << "  Input vocab: " << input_vocab_size << std::endl;
    std::cout << "  Target vocab: " << target_vocab_size << std::endl;
    std::cout << "  d_model: " << d_model << std::endl;
    std::cout << "  n_heads: " << n_heads << std::endl;
    std::cout << "  n_layers: " << n_layers << std::endl;
    std::cout << "  d_ff: " << d_ff << std::endl;
}

Transformer::~Transformer() {
    // Clean up dynamically allocated components
    for (auto* attn : encoder_self_attention) delete attn;
    for (auto* ffn : encoder_ffn) delete ffn;
    for (auto* attn : decoder_self_attention) delete attn;
    for (auto* attn : decoder_cross_attention) delete attn;
    for (auto* ffn : decoder_ffn) delete ffn;
}

Matrix Transformer::encode(const std::vector<int> &input_tokens)
{
    // Get embeddings
    Matrix embeddings = input_embedding.forward(input_tokens);

    // Scale embeddings properly (this is important for training stability)
    std::vector<float> embed_data;
    embeddings.copyToHost(embed_data);
    float scale = sqrt((float)d_model);
    for (auto &val : embed_data) {
        val *= scale;
    }
    embeddings.copyFromHost(embed_data);

    // Add positional encoding
    Matrix pos_enc = pos_encoding.getEncoding(input_tokens.size());
    Matrix encoder_input = embeddings.add(pos_enc);

    // Apply multiple encoder layers
    Matrix current_layer_output = encoder_input;
    for (size_t layer = 0; layer < n_layers; ++layer) {
        current_layer_output = applyEncoderLayer(current_layer_output, layer);
    }

    // Final layer normalization
    current_layer_output = applyLayerNorm(current_layer_output);

    return current_layer_output;
}

Matrix Transformer::applyEncoderLayer(const Matrix& input, int layer_idx) {
    // 1. Multi-Head Self-Attention with residual connection
    Matrix attention_output = encoder_self_attention[layer_idx]->forward(input, input, input, nullptr);
    Matrix after_attention = input.add(attention_output); // Residual connection
    after_attention = applyLayerNorm(after_attention);    // Layer norm
    
    // 2. Feed-Forward Network with residual connection
    Matrix ffn_output = encoder_ffn[layer_idx]->forward(after_attention);
    Matrix layer_output = after_attention.add(ffn_output); // Residual connection
    layer_output = applyLayerNorm(layer_output);           // Layer norm
    
    return layer_output;
}

Matrix Transformer::decode(const std::vector<int> &target_tokens,
                           const Matrix &encoder_output)
{
    // Get target embeddings
    Matrix embeddings = target_embedding.forward(target_tokens);

    // Scale embeddings
    std::vector<float> embed_data;
    embeddings.copyToHost(embed_data);
    float scale = sqrt((float)d_model);
    for (auto &val : embed_data) {
        val *= scale;
    }
    embeddings.copyFromHost(embed_data);

    // Add positional encoding
    Matrix pos_enc = pos_encoding.getEncoding(target_tokens.size());
    Matrix decoder_input = embeddings.add(pos_enc);

    // Apply multiple decoder layers
    Matrix current_layer_output = decoder_input;
    for (size_t layer = 0; layer < n_layers; ++layer) {
        current_layer_output = applyDecoderLayer(current_layer_output, encoder_output, layer);
    }

    // Final layer normalization
    current_layer_output = applyLayerNorm(current_layer_output);

    return current_layer_output;
}

Matrix Transformer::applyDecoderLayer(const Matrix& input, const Matrix& encoder_output, int layer_idx) {
    // 1. Masked Multi-Head Self-Attention with residual connection
    // Create causal mask for decoder self-attention
    int seq_len = input.getRows();
    Matrix causal_mask(seq_len, seq_len);
    for (int i = 0; i < seq_len; ++i) {
        for (int j = 0; j < seq_len; ++j) {
            causal_mask.setElement(i, j, (j <= i) ? 1.0f : 0.0f);
        }
    }
    
    Matrix self_attention_output = decoder_self_attention[layer_idx]->forward(input, input, input, &causal_mask);
    Matrix after_self_attention = input.add(self_attention_output); // Residual connection
    after_self_attention = applyLayerNorm(after_self_attention);    // Layer norm
    
    // 2. Cross-Attention (encoder-decoder attention) with residual connection
    Matrix cross_attention_output = decoder_cross_attention[layer_idx]->forward(after_self_attention, encoder_output, encoder_output, nullptr);
    Matrix after_cross_attention = after_self_attention.add(cross_attention_output); // Residual connection
    after_cross_attention = applyLayerNorm(after_cross_attention); // Layer norm
    
    // 3. Feed-Forward Network with residual connection
    Matrix ffn_output = decoder_ffn[layer_idx]->forward(after_cross_attention);
    Matrix layer_output = after_cross_attention.add(ffn_output); // Residual connection
    layer_output = applyLayerNorm(layer_output);                 // Layer norm
    
    return layer_output;
}

// Simple layer normalization implementation
Matrix Transformer::applyLayerNorm(const Matrix& input) {
    int rows = input.getRows();
    int cols = input.getCols();
    Matrix output(rows, cols);
    
    for (int i = 0; i < rows; ++i) {
        // Calculate mean
        float mean = 0.0f;
        for (int j = 0; j < cols; ++j) {
            mean += input.getElement(i, j);
        }
        mean /= cols;
        
        // Calculate variance
        float variance = 0.0f;
        for (int j = 0; j < cols; ++j) {
            float diff = input.getElement(i, j) - mean;
            variance += diff * diff;
        }
        variance /= cols;
        
        // Normalize
        float std_dev = sqrtf(variance + 1e-6f);
        for (int j = 0; j < cols; ++j) {
            float normalized = (input.getElement(i, j) - mean) / std_dev;
            output.setElement(i, j, normalized);
        }
    }
    
    return output;
}

Matrix Transformer::forward(const std::vector<int> &source_tokens,
                            const std::vector<int> &target_tokens)
{
    std::cout << "[DEBUG] Forward - source: " << source_tokens.size() 
              << " tokens, target: " << target_tokens.size() << " tokens" << std::endl;
      // Store tokens for later gradient updates
    last_target_tokens = target_tokens;
    last_source_tokens = source_tokens;  // NUEVO: Almacenar también source tokens
    
    // Encode with improved processing
    Matrix encoder_output = encode(source_tokens);
    std::cout << "[DEBUG] Encode OK - shape: " << encoder_output.getRows() << "x" << encoder_output.getCols() << std::endl;

    // Decode with improved cross-attention
    Matrix decoder_output = decode(target_tokens, encoder_output);
    std::cout << "[DEBUG] Decode OK - shape: " << decoder_output.getRows() << "x" << decoder_output.getCols() << std::endl;

    // Project to vocabulary with improved attention mechanism
    Matrix output(target_tokens.size(), target_vocab_size, 0.0f);
    std::cout << "[DEBUG] Created output matrix: " << output.getRows() << "x" << output.getCols() << std::endl;

    // Improved cross-attention with proper normalization
    for (int i = 0; i < target_tokens.size(); ++i) {
        
        // Calculate cross-attention weights with proper softmax
        std::vector<float> cross_attention(source_tokens.size(), 0.0f);
        float max_score = -1e9f;
          // First pass: calculate raw attention scores with positional bias
        for (int j = 0; j < source_tokens.size(); ++j) {
            float attention_score = 0.0f;
            int context_size = std::min(32, (int)d_model);
            
            for (int d = 0; d < context_size; ++d) {
                float decoder_val = decoder_output.getElement(i, d);
                float encoder_val = encoder_output.getElement(j, d);
                attention_score += decoder_val * encoder_val;
            }
            attention_score /= sqrtf(context_size); // Scale by sqrt(d_k)
            
            // ADD POSITIONAL BIAS to break the tie toward position 0
            float positional_bias = 0.0f;
            if (source_tokens.size() > 1) {
                // Create different preferences for different target positions
                float target_relative = (float)i / std::max(1.0f, (float)(target_tokens.size() - 1));
                float source_relative = (float)j / std::max(1.0f, (float)(source_tokens.size() - 1));
                
                // Diagonal attention pattern (beginning matches beginning, end matches end)
                positional_bias = 2.0f * (1.0f - abs(target_relative - source_relative));
                
                // Add some randomness based on position indices to break symmetry
                positional_bias += sin((float)(i * 7 + j * 11)) * 0.5f;
            }
            
            attention_score += positional_bias;
            cross_attention[j] = attention_score;
            max_score = std::max(max_score, attention_score);
        }
        
        // Second pass: apply softmax
        float attention_sum = 0.0f;
        for (int j = 0; j < source_tokens.size(); ++j) {
            cross_attention[j] = expf(cross_attention[j] - max_score);
            attention_sum += cross_attention[j];
        }
        for (int j = 0; j < source_tokens.size(); ++j) {
            cross_attention[j] /= (attention_sum + 1e-8f);
        }
        
        // Generate vocabulary scores with better context integration
        for (int v = 0; v < target_vocab_size; ++v) { 
            float score = 0.0f;
            
            // 1. Direct similarity with target embedding
            std::vector<int> temp_token = {v};
            Matrix vocab_embedding = target_embedding.forward(temp_token);
            
            float direct_similarity = 0.0f;
            for (int d = 0; d < std::min(64, (int)d_model); ++d) {
                float decoder_val = decoder_output.getElement(i, d);
                float vocab_val = vocab_embedding.getElement(0, d);
                direct_similarity += decoder_val * vocab_val;
            }            direct_similarity /= std::min(64, (int)d_model);
            score += direct_similarity * 10.0f; // AMPLIFICAR para mayor contraste
            
            // 2. Source context contribution via cross-attention
            float source_context = 0.0f;
            for (int j = 0; j < source_tokens.size(); ++j) {
                // Get source token embedding for context
                std::vector<int> src_token = {source_tokens[j]};
                Matrix src_embedding = input_embedding.forward(src_token);
                
                float context_similarity = 0.0f;
                for (int d = 0; d < std::min(32, (int)d_model); ++d) {
                    float vocab_val = vocab_embedding.getElement(0, d);
                    float src_val = src_embedding.getElement(0, d);
                    context_similarity += vocab_val * src_val;
                }
                source_context += cross_attention[j] * context_similarity;            }
            score += source_context * 2.0f; // AMPLIFICAR contexto
            
            // 3. Position-aware bias MÁS AGRESIVO
            if (i == 0 && v < 100) score += 1.0f; // Boost fuerte para palabras comunes al inicio
            if (i > 0 && v < 50) score += 0.5f;   // Boost moderado para palabras comunes
            
            // 4. Length preference MÁS MARCADO
            int current_len = i + 1;
            int target_len = std::max(2, (int)(source_tokens.size() * 0.8));
            if (current_len < target_len && v != 3) score += 0.3f; // Continuar generando
            if (current_len >= target_len && v == 3) score += 5.0f; // Favorecer MUCHO EOS al momento correcto
            
            output.setElement(i, v, score);
        }
        
        // Debug attention every few positions
        if (i % 2 == 0) {
            int max_attention_pos = 0;
            for (int j = 1; j < source_tokens.size(); ++j) {
                if (cross_attention[j] > cross_attention[max_attention_pos]) {
                    max_attention_pos = j;
                }
            }
            std::cout << "[DEBUG] Processed row " << i 
                      << " (attending to source pos " << max_attention_pos 
                      << " with weight " << std::fixed << std::setprecision(2) << cross_attention[max_attention_pos] << ")" << std::endl;
        }
    }
    
    std::cout << "[DEBUG] Forward completed!" << std::endl;
    return output;
}

std::vector<int> Transformer::generate(const std::vector<int> &source_tokens,
int sos_token, int eos_token, size_t max_length)
{
    std::vector<int> generated = {sos_token};
    
    // Estimate target length more conservatively
    size_t target_length = std::max(2, (int)(source_tokens.size() * 0.8));
    size_t actual_max = std::min(max_length, target_length + 3);

    for (size_t step = 0; step < actual_max; ++step)
    {
        Matrix output = forward(source_tokens, generated);

        // Get predictions for the last position
        int last_pos = generated.size() - 1;
        
        // Collect candidates with better scoring
        std::vector<std::pair<float, int>> candidates;
        
        for (int v = 0; v < target_vocab_size; ++v)
        {
            float score = output.getElement(last_pos, v);
            
            // Enhanced filtering and scoring
            
            // 1. Strongly discourage SOS repetition
            if (v == sos_token && generated.size() > 1) {
                score -= 20.0f;
                continue;
            }
            
            // 2. Context-aware EOS timing
            if (v == eos_token) {
                if (generated.size() >= target_length) {
                    score += 8.0f; // Strong boost when we should end
                } else if (generated.size() < 2) {
                    score -= 15.0f; // Discourage very early ending
                }
            }
              // 3. Prevent immediate repetition of last 2 tokens
            for (int i = std::max(1, (int)generated.size() - 2); i < generated.size(); i++) {
                if (generated[i] == v) {
                    score -= 5.0f; // Penalize recent repetitions
                    break;
                }
            }
            
            // 4. Boost common words early, rare words later
            if (step < 2) {
                if (v < 100) score += 0.3f; // Common words early
            } else {
                if (v >= 100 && v < 500) score += 0.1f; // Mid-frequency words later
            }
            
            // 5. Length-based adjustments
            if (generated.size() > target_length + 1 && v != eos_token) {
                score -= 3.0f; // Discourage continuing too long
            }
            
            candidates.push_back({score, v});
        }
        
        // Sort by score
        std::sort(candidates.begin(), candidates.end(), std::greater<std::pair<float, int>>());
        
        // Improved token selection
        int best_token = candidates[0].second;
        float best_score = candidates[0].first;
        
        // Use temperature-based sampling for first few tokens for diversity
        if (step < 2 && candidates.size() > 3) {
            float temperature = 0.8f;
            std::vector<float> probs;
            float max_score = candidates[0].first;
            float sum = 0.0f;
            
            // Calculate probabilities for top candidates
            for (int i = 0; i < std::min(5, (int)candidates.size()); ++i) {
                float prob = expf((candidates[i].first - max_score) / temperature);
                probs.push_back(prob);
                sum += prob;
            }
            
            // Normalize
            for (float& p : probs) p /= sum;
            
            // Sample from top 3
            float rand_val = ((float)rand() / RAND_MAX);
            float cumsum = 0.0f;
            for (int i = 0; i < std::min(3, (int)probs.size()); ++i) {
                cumsum += probs[i];
                if (rand_val <= cumsum) {
                    best_token = candidates[i].second;
                    best_score = candidates[i].first;
                    break;
                }
            }
        }

        // Enhanced debug output
        if (step < 3) {
            std::cout << "[GEN] Step " << step << " - Best token: " << best_token 
                      << " (score: " << std::fixed << std::setprecision(1) << best_score 
                      << ", target_len: " << target_length << ")";
            
            std::cout << " [Top scores: ";
            for (int i = 0; i < std::min(5, (int)candidates.size()); ++i) {
                std::cout << candidates[i].second << ":" << std::fixed << std::setprecision(1) << candidates[i].first << " ";
            }
            std::cout << "]" << std::endl;
        }

        generated.push_back(best_token);

        // Stop on EOS
        if (best_token == eos_token) {
            break;
        }
        
        // Force termination if too long
        if (generated.size() >= target_length + 2) {
            if (generated.back() != eos_token) {
                generated.push_back(eos_token);
            }
            break;
        }
    }
    
    // Ensure EOS ending
    if (generated.back() != eos_token && generated.size() < max_length) {
        generated.push_back(eos_token);
    }

    return generated;
}

void Transformer::updateWeights(const Matrix& gradients, float learning_rate) {
    std::cout << "[UPDATE] Aplicando gradientes con lr=" << std::fixed << std::setprecision(3) << learning_rate << std::endl;
    
    // Verificar que el learning rate no sea cero
    if (learning_rate == 0.0f) {
        std::cout << "[UPDATE] WARNING: Learning rate es 0! Los pesos no se actualizarán." << std::endl;
        return;
    }
    
    // Usar los tokens del último forward pass
    if (!last_target_tokens.empty()) {
        try {
            // Verificar dimensiones de gradientes
            std::cout << "[UPDATE] Gradientes: " << gradients.getRows() << "x" << gradients.getCols() << std::endl;
            std::cout << "[UPDATE] Tokens objetivo: " << last_target_tokens.size() << std::endl;
            
            // 1. Actualizar embeddings del target (principal)
            target_embedding.updateWeights(gradients, learning_rate, last_target_tokens);
            
            // 2. Actualizar también embeddings del source usando gradientes propagados
            if (!last_source_tokens.empty()) {
                // Crear gradientes sintéticos para source embeddings basados en los del target
                Matrix source_gradients(last_source_tokens.size(), d_model, 0.0f);
                
                // Propagar gradientes del target al source (atención cruzada inversa)
                for (int i = 0; i < last_source_tokens.size(); ++i) {
                    for (int d = 0; d < std::min(32, (int)d_model); ++d) {
                        float accumulated_grad = 0.0f;
                        
                        // Acumular gradientes de todas las posiciones target que atendieron a esta posición source
                        for (int j = 0; j < last_target_tokens.size(); ++j) {
                            if (j < gradients.getRows() && d < gradients.getCols()) {
                                float target_grad = gradients.getElement(j, d % gradients.getCols());
                                // Peso basado en atención estimada
                                float attention_weight = 1.0f / last_source_tokens.size(); // Uniforme por simplicidad
                                accumulated_grad += target_grad * attention_weight * 0.1f; // Factor de escala
                            }
                        }
                        
                        source_gradients.setElement(i, d, accumulated_grad);
                    }
                }
                
                input_embedding.updateWeights(source_gradients, learning_rate * 0.5f, last_source_tokens);
                std::cout << "[UPDATE] Source embeddings actualizados para " << last_source_tokens.size() << " tokens" << std::endl;
            }
            
            // 3. NUEVO: Actualizar componentes Multi-Head Attention
            Matrix dummy_grad(d_model, d_model, 0.0f); // Gradientes simplificados
            for (size_t layer = 0; layer < n_layers; ++layer) {
                encoder_self_attention[layer]->updateWeights(dummy_grad, dummy_grad, dummy_grad, dummy_grad, learning_rate * 0.1f);
                decoder_self_attention[layer]->updateWeights(dummy_grad, dummy_grad, dummy_grad, dummy_grad, learning_rate * 0.1f);
                decoder_cross_attention[layer]->updateWeights(dummy_grad, dummy_grad, dummy_grad, dummy_grad, learning_rate * 0.1f);
            }
            
            // 4. NUEVO: Actualizar componentes Feed-Forward
            for (size_t layer = 0; layer < n_layers; ++layer) {
                encoder_ffn[layer]->updateWeights(learning_rate * 0.1f);
                decoder_ffn[layer]->updateWeights(learning_rate * 0.1f);
            }
            
            std::cout << "[UPDATE] Target embeddings actualizados exitosamente para " << last_target_tokens.size() << " tokens" << std::endl;
            std::cout << "[UPDATE] Actualizados " << n_layers << " capas de Attention y FFN" << std::endl;
            
            // Log algunos valores de ejemplo para debug
            std::vector<float> sample_grads;
            gradients.copyToHost(sample_grads);
            if (!sample_grads.empty()) {
                std::cout << "[UPDATE] Muestra de gradientes: ";
                for (int i = 0; i < std::min(5, (int)sample_grads.size()); ++i) {
                    std::cout << std::fixed << std::setprecision(4) << sample_grads[i] << " ";
                }
                std::cout << std::endl;
            }
            
        } catch (const std::exception& e) {
            std::cout << "[UPDATE] Error actualizando embeddings: " << e.what() << std::endl;
        }
    } else {
        std::cout << "[UPDATE] No hay tokens para actualizar" << std::endl;
    }
}