#include "hip/hip_runtime.h"
#include "transformer.cuh"
#include "embeddings.cuh"
#include "../utils/matrix.cuh"
#include <iostream>
#include <cmath>

Transformer::Transformer(size_t input_vocab_size, size_t target_vocab_size,size_t d_model, size_t n_heads, size_t n_layers, size_t d_ff)
    : input_vocab_size(input_vocab_size), target_vocab_size(target_vocab_size),
      d_model(d_model), n_layers(n_layers),
      input_embedding(input_vocab_size, d_model),
      target_embedding(target_vocab_size, d_model),
      pos_encoding(d_model)
{

    std::cout << "Transformer initialized:" << std::endl;
    std::cout << "  Input vocab: " << input_vocab_size << std::endl;
    std::cout << "  Target vocab: " << target_vocab_size << std::endl;
    std::cout << "  d_model: " << d_model << std::endl;
    std::cout << "  layers: " << n_layers << std::endl;
}

Matrix Transformer::encode(const std::vector<int> &input_tokens)
{
    // Get embeddings
    Matrix embeddings = input_embedding.forward(input_tokens);

    // Scale embeddings
    std::vector<float> embed_data;
    embeddings.copyToHost(embed_data);
    float scale = sqrt(d_model);
    for (auto &val : embed_data)
    {
        val *= scale;
    }
    embeddings.copyFromHost(embed_data);

    // Add positional encoding
    Matrix pos_enc = pos_encoding.getEncoding(input_tokens.size());
    Matrix encoder_input = embeddings.add(pos_enc);

    // For now, return encoder_input (no actual encoder layers yet)
    return encoder_input;
}

Matrix Transformer::decode(const std::vector<int> &target_tokens,
                           const Matrix &encoder_output)
{
    // Get target embeddings
    Matrix embeddings = target_embedding.forward(target_tokens);

    // Scale embeddings
    std::vector<float> embed_data;
    embeddings.copyToHost(embed_data);
    float scale = sqrt(d_model);
    for (auto &val : embed_data)
    {
        val *= scale;
    }
    embeddings.copyFromHost(embed_data);

    // Add positional encoding
    Matrix pos_enc = pos_encoding.getEncoding(target_tokens.size());
    Matrix decoder_input = embeddings.add(pos_enc);

    // For now, return decoder_input (no actual decoder layers yet)
    return decoder_input;
}

Matrix Transformer::forward(const std::vector<int> &source_tokens,
                            const std::vector<int> &target_tokens)
{
    std::cout << "[DEBUG] Forward - source: " << source_tokens.size() 
              << " tokens, target: " << target_tokens.size() << " tokens" << std::endl;
    
    // Encode
    Matrix encoder_output = encode(source_tokens);
    std::cout << "[DEBUG] Encode OK - shape: " << encoder_output.getRows() << "x" << encoder_output.getCols() << std::endl;

    // Decode
    Matrix decoder_output = decode(target_tokens, encoder_output);
    std::cout << "[DEBUG] Decode OK - shape: " << decoder_output.getRows() << "x" << decoder_output.getCols() << std::endl;

    // Project to vocabulary (simplified linear projection)
    Matrix output(target_tokens.size(), target_vocab_size, 0.0f);
    std::cout << "[DEBUG] Created output matrix: " << output.getRows() << "x" << output.getCols() << std::endl;

    // SIMPLIFICA LA PROYECCIÓN - Hazla más rápida
    for (int i = 0; i < target_tokens.size(); ++i) {
        for (int v = 0; v < std::min(100, (int)target_vocab_size); ++v) { // Solo 100 clases por ahora
            output.setElement(i, v, 0.1f * (v + 1)); // Valores dummy simples
        }
        if (i % 2 == 0) {
            std::cout << "[DEBUG] Processed row " << i << std::endl;
        }
    }
    
    std::cout << "[DEBUG] Forward completed!" << std::endl;
    return output;
}

std::vector<int> Transformer::generate(const std::vector<int> &source_tokens,
int sos_token, int eos_token, size_t max_length)
{
    std::vector<int> generated = {sos_token};

    for (size_t step = 0; step < max_length; ++step)
    {
        Matrix output = forward(source_tokens, generated);

        // Get last token predictions
        int last_pos = generated.size() - 1;
        int best_token = 0;
        float best_score = output.getElement(last_pos, 0);

        for (int v = 1; v < target_vocab_size; ++v)
        {
            float score = output.getElement(last_pos, v);
            if (score > best_score)
            {
                best_score = score;
                best_token = v;
            }
        }

        generated.push_back(best_token);

        if (best_token == eos_token)
        {
            break;
        }
    }

    return generated;
}