#include "hip/hip_runtime.h"
// filepath: cuda-transformer/cuda-transformer/src/transformer/encoder.cu
#include "encoder.cuh"
#include "attention.cuh"
#include "layers/feed_forward.cuh"
#include "layers/layer_norm.cuh"
#include "utils/cuda_utils.cuh"

Matrix EncoderLayer::forward(const Matrix &input, const Matrix *src_mask) {
    // Self-attention
    Matrix self_att_output = self_attention.forward(input, input, input, src_mask ? *src_mask : Matrix());
    Matrix norm1_output = norm1.forward(input.add(self_att_output));

    // Feed-forward
    Matrix ff_output = feed_forward.forward(norm1_output);
    Matrix norm2_output = norm2.forward(norm1_output.add(ff_output));

    return norm2_output;
}


// Kernel function to launch multiple encoder layers
__global__ void encodeKernel(Matrix *input, Matrix *output, Matrix *src_mask, EncoderLayer *layers, size_t n_layers) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_layers) {
        output[idx] = layers[idx].forward(input[idx], &src_mask[idx]);
    }
}

void Encoder::forward(const Matrix &input, const Matrix &src_mask, Matrix &output) {
    // Launch kernel for encoding
    int blockSize = 256;
    int numBlocks = (n_layers + blockSize - 1) / blockSize;
    encodeKernel<<<numBlocks, blockSize>>>(input.device_ptr(), output.device_ptr(), src_mask.device_ptr(), layers.device_ptr(), n_layers);
    hipDeviceSynchronize();
}