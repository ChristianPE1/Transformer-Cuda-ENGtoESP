#include "hip/hip_runtime.h"
// src/utils/matrix.cu
#include "matrix.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixAddKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        C[row * cols + col] = A[row * cols + col] + B[row * cols + col];
    }
}

Matrix::Matrix(int rows, int cols) : rows(rows), cols(cols) {
    hipMalloc(&data, rows * cols * sizeof(float));
}

Matrix::~Matrix() {
    hipFree(data);
}

void Matrix::add(const Matrix& other, Matrix& result) {
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);
    matrixAddKernel<<<gridSize, blockSize>>>(data, other.data, result.data, rows, cols);
    hipDeviceSynchronize();
}

void Matrix::copyFromHost(const float* hostData) {
    hipMemcpy(data, hostData, rows * cols * sizeof(float), hipMemcpyHostToDevice);
}

void Matrix::copyToHost(float* hostData) const {
    hipMemcpy(hostData, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
}

void Matrix::initialize(float value) {
    float* hostData = new float[rows * cols];
    for (int i = 0; i < rows * cols; ++i) {
        hostData[i] = value;
    }
    copyFromHost(hostData);
    delete[] hostData;
}

int Matrix::getRows() const {
    return rows;
}

int Matrix::getCols() const {
    return cols;
}