#include "hip/hip_runtime.h"
// src/training/optimizer.cu
#include "optimizer.cuh"
#include "utils/cuda_utils.cuh"

__global__ void updateWeightsKernel(float *weights, float *gradients, float learning_rate, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        weights[idx] -= learning_rate * gradients[idx];
    }
}

Optimizer::Optimizer(float learning_rate) : learning_rate(learning_rate) {}

void Optimizer::update(float *weights, float *gradients, int size) {
    float *d_weights, *d_gradients;

    hipMalloc(&d_weights, size * sizeof(float));
    hipMalloc(&d_gradients, size * sizeof(float));

    hipMemcpy(d_weights, weights, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gradients, gradients, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    updateWeightsKernel<<<numBlocks, blockSize>>>(d_weights, d_gradients, learning_rate, size);

    hipMemcpy(weights, d_weights, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_weights);
    hipFree(d_gradients);
}